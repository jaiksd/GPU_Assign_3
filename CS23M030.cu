#include "hip/hip_runtime.h"
#include "SceneNode.h"
#include <queue>
#include "Renderer.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <chrono>


__global__ void parallelTraslation(int* deviceChangeX, int* deviceChangeY, int* trn, int* trc, int* tra, int T)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= T)
		return;
	else {
		if (trc[idx] == 0) atomicSub(&deviceChangeX[trn[idx]], tra[idx]);
		else if (trc[idx] == 1) atomicAdd(&deviceChangeX[trn[idx]], tra[idx]);
		else if (trc[idx] == 2) atomicSub(&deviceChangeY[trn[idx]], tra[idx]);
		else if (trc[idx] == 3)atomicAdd(&deviceChangeY[trn[idx]], tra[idx]);
	}
}





__global__ void update_level(int* deviceChangeX, int* deviceChangeY, int* deviceGlobalCoordinatesX, int* deviceGlobalCoordinatesY, int* queue, int qStart, int qEnd, int* qptr, int* deviceCsr, int* deviceOffset)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid + qStart >= qEnd)
		return;
	else {
		deviceGlobalCoordinatesY[queue[tid + qStart]] += deviceChangeY[queue[tid + qStart]];
		deviceGlobalCoordinatesX[queue[tid + qStart]] += deviceChangeX[queue[tid + qStart]];
		int newStart = atomicAdd(qptr, deviceOffset[queue[tid + qStart] + 1] - deviceOffset[queue[tid + qStart]]);
		int i = deviceOffset[queue[tid + qStart]];
		while (i < deviceOffset[queue[tid + qStart] + 1])
		{
			deviceChangeX[deviceCsr[i]] += deviceChangeX[queue[tid + qStart]];
			deviceChangeY[deviceCsr[i]] += deviceChangeY[queue[tid + qStart]];
			queue[newStart + i - deviceOffset[queue[tid + qStart]]] = deviceCsr[i];
			i++;
		}
	}
}

__global__ void finalWrite(int node, int* gjpeg, int* gop, int* devicemesh, int meshop, int* gX, int* gY, int meshY, int frameX, int frameY)
{

	int fr = blockIdx.x + gX[node];
	int fc = threadIdx.x + gY[node];
	if (fr >= frameX || fc >= frameY || fr < 0 || fc < 0)
		return;
	else
	{
		if (meshop < gop[(fr)*frameY + fc])
			return;
		else
		{
			gop[(fr)*frameY + fc] = meshop;
			gjpeg[(fr)*frameY + fc] = devicemesh[blockIdx.x * meshY + threadIdx.x];
		}

	}
}

void readFile(const char* fileName, std::vector<SceneNode*>& scenes, std::vector<std::vector<int> >& edges, std::vector<std::vector<int> >& translations, int& frameSizeX, int& frameSizeY) {
	/* Function for parsing input file*/

	FILE* inputFile = NULL;
	// Read the file for input. 
	if ((inputFile = fopen(fileName, "r")) == NULL) {
		printf("Failed at opening the file %s\n", fileName);
		return;
	}

	// Input the header information.
	int numMeshes;
	fscanf(inputFile, "%d", &numMeshes);
	fscanf(inputFile, "%d %d", &frameSizeX, &frameSizeY);


	// Input all meshes and store them inside a vector.
	int meshX, meshY;
	int globalPositionX, globalPositionY; // top left corner of the matrix.
	int opacity;
	int* currMesh;
	for (int i = 0; i < numMeshes; i++) {
		fscanf(inputFile, "%d %d", &meshX, &meshY);
		fscanf(inputFile, "%d %d", &globalPositionX, &globalPositionY);
		fscanf(inputFile, "%d", &opacity);
		currMesh = (int*)malloc(sizeof(int) * meshX * meshY);
		for (int j = 0; j < meshX; j++) {
			for (int k = 0; k < meshY; k++) {
				fscanf(inputFile, "%d", &currMesh[j * meshY + k]);
			}
		}
		//Create a Scene out of the mesh.
		SceneNode* scene = new SceneNode(i, currMesh, meshX, meshY, globalPositionX, globalPositionY, opacity);
		scenes.push_back(scene);
	}

	// Input all relations and store them in edges.
	int relations;
	fscanf(inputFile, "%d", &relations);
	int u, v;
	for (int i = 0; i < relations; i++) {
		fscanf(inputFile, "%d %d", &u, &v);
		edges.push_back({ u,v });
	}

	// Input all translations.
	int numTranslations;
	fscanf(inputFile, "%d", &numTranslations);
	std::vector<int> command(3, 0);
	for (int i = 0; i < numTranslations; i++) {
		fscanf(inputFile, "%d %d %d", &command[0], &command[1], &command[2]);
		translations.push_back(command);
	}
}


void writeFile(const char* outputFileName, int* hFinalPng, int frameSizeX, int frameSizeY) {
	/* Function for writing the final png into a file.*/
	FILE* outputFile = NULL;
	if ((outputFile = fopen(outputFileName, "w")) == NULL) {
		printf("Failed while opening output file\n");
	}

	for (int i = 0; i < frameSizeX; i++) {
		for (int j = 0; j < frameSizeY; j++) {
			fprintf(outputFile, "%d ", hFinalPng[i * frameSizeY + j]);
		}
		fprintf(outputFile, "\n");
	}
}


int main(int argc, char** argv) {

	// Read the scenes into memory from File.
	const char* inputFileName = argv[1];
	int* hFinalPng;

	int frameSizeX, frameSizeY;
	std::vector<SceneNode*> scenes;
	std::vector<std::vector<int> > edges;
	std::vector<std::vector<int> > translations;
	readFile(inputFileName, scenes, edges, translations, frameSizeX, frameSizeY);
	hFinalPng = (int*)malloc(sizeof(int) * frameSizeX * frameSizeY);

	// Make the scene graph from the matrices.
	Renderer* scene = new Renderer(scenes, edges);

	// Basic information.
	int V = scenes.size();
	int E = edges.size();
	int numTranslations = translations.size();

	// Convert the scene graph into a csr.
	scene->make_csr(); // Returns the Compressed Sparse Row representation for the graph.
	int* hOffset = scene->get_h_offset();
	int* hCsr = scene->get_h_csr();
	int* hOpacity = scene->get_opacity(); // hOpacity[vertexNumber] contains opacity of vertex vertexNumber.
	int** hMesh = scene->get_mesh_csr(); // hMesh[vertexNumber] contains the mesh attached to vertex vertexNumber.
	int* hGlobalCoordinatesX = scene->getGlobalCoordinatesX(); // hGlobalCoordinatesX[vertexNumber] contains the X coordinate of the vertex vertexNumber.
	int* hGlobalCoordinatesY = scene->getGlobalCoordinatesY(); // hGlobalCoordinatesY[vertexNumber] contains the Y coordinate of the vertex vertexNumber.
	int* hFrameSizeX = scene->getFrameSizeX(); // hFrameSizeX[vertexNumber] contains the vertical size of the mesh attached to vertex vertexNumber.
	int* hFrameSizeY = scene->getFrameSizeY(); // hFrameSizeY[vertexNumber] contains the horizontal size of the mesh attached to vertex vertexNumber.

	auto start = std::chrono::high_resolution_clock::now();


	// Code begins here.
	// Do not change anything above this comment.
	int* deviceCsr;
	hipMalloc(&deviceCsr, sizeof(int) * E);
	hipMemcpy(deviceCsr, hCsr, sizeof(int) * E, hipMemcpyHostToDevice);

	int* deviceOffset;
	hipMalloc(&deviceOffset, sizeof(int) * (V + 1));
	hipMemcpy(deviceOffset, hOffset, sizeof(int) * (V + 1), hipMemcpyHostToDevice);

	int* DeviceGlobalCoordinatesX;

	int htrn[numTranslations], htrc[numTranslations], htra[numTranslations];
	int i = 0;

	hipMalloc(&DeviceGlobalCoordinatesX, sizeof(int) * (V));
	hipMemcpy(DeviceGlobalCoordinatesX, hGlobalCoordinatesX, sizeof(int) * V, hipMemcpyHostToDevice);
	int* deviceChangeX;
	hipMalloc(&deviceChangeX, sizeof(int) * V);
	hipMemset(deviceChangeX, 0, sizeof(int) * V);

	int* deviceChangeY;
	hipMalloc(&deviceChangeY, sizeof(int) * V);
	hipMemset(deviceChangeY, 0, sizeof(int) * V);


	int* deviceGlobalCoordinatesY;
	hipMalloc(&deviceGlobalCoordinatesY, sizeof(int) * (V));
	hipMemcpy(deviceGlobalCoordinatesY, hGlobalCoordinatesY, sizeof(int) * V, hipMemcpyHostToDevice);

	while (i < numTranslations)
	{
		htrn[i] = translations[i][0];
		htrc[i] = translations[i][1];
		htra[i] = translations[i][2];
		i++;
	}

	int* trn, * trc, * tra;
	hipMalloc(&trn, sizeof(int) * numTranslations);
	hipMemcpy(trn, htrn, sizeof(int) * numTranslations, hipMemcpyHostToDevice);

	hipMalloc(&trc, sizeof(int) * numTranslations);
	hipMemcpy(trc, htrc, sizeof(int) * numTranslations, hipMemcpyHostToDevice);
	hipMalloc(&tra, sizeof(int) * numTranslations);
	hipMemcpy(tra, htra, sizeof(int) * numTranslations, hipMemcpyHostToDevice);

	int numblocks_kernel1 = ceil(1.0 * numTranslations / 1024);
	int numthreadsperblock = 1024;
	if (numblocks_kernel1 == 0)
	{
		numblocks_kernel1 = 1;
		numthreadsperblock = numTranslations;
	}
	parallelTraslation << <numblocks_kernel1, numthreadsperblock >> > (deviceChangeX, deviceChangeY, trn, trc, tra, numTranslations);
	hipError_t err1 = hipGetLastError();
	hipDeviceSynchronize();

	hipFree(tra);
	hipFree(trc);
	hipFree(trn);

	int* gqueue;
	hipMalloc(&gqueue, V * sizeof(int));
	hipMemset(gqueue, 0, sizeof(int) * V);
	int* hqueuetop;
	hqueuetop = (int*)malloc(sizeof(int));
	*hqueuetop = 1;


	int* gqueuetop;
	hipMalloc(&gqueuetop, sizeof(int));
	hipMemcpy(gqueuetop, hqueuetop, sizeof(int), hipMemcpyHostToDevice);

	int* hqueueback;
	hqueueback = (int*)malloc(sizeof(int));
	*hqueueback = 0;

	while ((*hqueuetop) - (*hqueueback) > 0)
	{
		update_level << < ((*hqueuetop - *hqueueback) + 1023) / 1024, 1024 >> > (deviceChangeX, deviceChangeY, DeviceGlobalCoordinatesX, deviceGlobalCoordinatesY, gqueue, *hqueueback, *hqueuetop, gqueuetop, deviceCsr, deviceOffset);
		hipDeviceSynchronize();
		*hqueueback = *hqueuetop;
		hipMemcpy(hqueuetop, gqueuetop, sizeof(int), hipMemcpyDeviceToHost);
	}
	hipDeviceSynchronize();

	hipFree(gqueue);
	hipFree(deviceCsr);
	hipFree(deviceChangeY);
	hipFree(gqueuetop);
	hipFree(deviceChangeX);
	hipFree(deviceOffset);

	int* gop;
	hipMalloc(&gop, sizeof(int) * frameSizeX * frameSizeY);
	hipMemset(gop, -1, sizeof(int) * frameSizeX * frameSizeY);

	int* devicemesh;
	hipMalloc(&devicemesh, sizeof(int) * 10000);
	hipMemset(devicemesh, 0, sizeof(int) * 10000);

	int* gjpeg;
	hipMalloc(&gjpeg, sizeof(int) * frameSizeX * frameSizeY);
	hipMemset(gjpeg, 0, sizeof(int) * frameSizeX * frameSizeY);


	int q = 0;
	while (q < V)
	{
		hipMemcpy(devicemesh, hMesh[q], sizeof(int) * hFrameSizeX[q] * hFrameSizeY[q], hipMemcpyHostToDevice);
		finalWrite << <hFrameSizeX[q], hFrameSizeY[q] >> > (q, gjpeg, gop, devicemesh, hOpacity[q], DeviceGlobalCoordinatesX, deviceGlobalCoordinatesY, hFrameSizeY[q], frameSizeX, frameSizeY);
		q++;
	}
	hipMemcpy(hFinalPng, gjpeg, sizeof(int) * frameSizeX * frameSizeY, hipMemcpyDeviceToHost);
	


	// Do not change anything below this comment.
	// Code ends here.

	auto end = std::chrono::high_resolution_clock::now();

	std::chrono::duration<double, std::micro> timeTaken = end - start;

	printf("execution time : %f\n", timeTaken);
	// Write output matrix to file.
	const char* outputFileName = argv[2];
	writeFile(outputFileName, hFinalPng, frameSizeX, frameSizeY);

}
